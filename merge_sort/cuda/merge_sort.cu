#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>
#include <stdlib.h>
#include <stdio.h>

int THREADS;
int BLOCKS;
int NUM_VALS;  

// Generate data
void generate_data(size_t size, int *data) {
    for (size_t i = 0; i < size; i++)
    {
        data[i] = rand() % (size * 10);
    }
}

// Correctness check
bool is_correct(size_t size, int *data) {
    for (size_t i = 1; i < size; i++) {
        if (data[i - 1] > data[i]) {
            return false;
        }
    }
    return true;
}

__device__ void deviceMerge(int *array, int *temp, int left, int right, int middle) {
    int left_idx = left;
    int merged_idx = left;
    int right_idx = middle+1;
    int k;

    // Sort left and right side of array into temp
    while ((left_idx <= middle) && (right_idx <= right)) {
        if (array[left_idx] <= array[right_idx]) {
            temp[merged_idx] = array[left_idx];
            left_idx++;
        } else {
            temp[merged_idx] = array[right_idx];
            right_idx++;
        }
        merged_idx++;
    }

    // Copy remaining elements into temp array
    if (left_idx > middle) {
        for (k=right_idx; k<=right; k++) {
            temp[merged_idx] = array[k];
            merged_idx++;
        }
    } else {
        for (k=left_idx; k<=middle; k++) {
            temp[merged_idx] = array[k];
            merged_idx++;
        }
    }

    // Put sorted temp back into array
    for (k=left; k<=right; k++) {
        array[k] = temp[k];
    }
}

__device__ void deviceMergeSort(int *array, int *temp, int left, int right) {
    if (left < right) {
        int middle = (left+right)/2;
        deviceMergeSort(array, temp, left, middle);
        deviceMergeSort(array, temp, middle+1, right);
        deviceMerge(array, temp, left, right, middle);
    }
}

__global__ void mergeSortKernel(int *data, int num_vals) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int threads = blockDim.x * gridDim.x;

    int size = num_vals / threads;
    int left = tid * size;
    int right = (tid + 1) * size - 1;

    int *temp = (int*) malloc(num_vals * sizeof(int));
    deviceMergeSort(data, temp, left, right);
    free(temp);
}

void merge(int *array, int *temp, int left, int right, int middle) {
    int left_idx = left;
    int merged_idx = left;
    int right_idx = middle+1;
    int k;

    // Sort left and right side of array into temp
    CALI_MARK_BEGIN("comp");
    CALI_MARK_BEGIN("comp_large");
    while ((left_idx <= middle) && (right_idx <= right)) {
        if (array[left_idx] <= array[right_idx]) {
            temp[merged_idx] = array[left_idx];
            left_idx++;
        } else {
            temp[merged_idx] = array[right_idx];
            right_idx++;
        }
        merged_idx++;
    }
    CALI_MARK_END("comp_large");
    CALI_MARK_END("comp");

    // Copy remaining elements into temp array
    CALI_MARK_BEGIN("comp");
    CALI_MARK_BEGIN("comp_small");
    if (left_idx > middle) {
        for (k=right_idx; k<=right; k++) {
            temp[merged_idx] = array[k];
            merged_idx++;
        }
    } else {
        for (k=left_idx; k<=middle; k++) {
            temp[merged_idx] = array[k];
            merged_idx++;
        }
    }
    CALI_MARK_END("comp_small");
    CALI_MARK_END("comp");

    // Put sorted temp back into array
    for (k=left; k<=right; k++) {
        array[k] = temp[k];
    }
}

void finalMerge(int *array, int *temp, int left, int right, int num_sub_arrays) {
    int middle = (left+right)/2;
    if (num_sub_arrays != 2) {
        //call final merge again
        finalMerge(array, temp, left, middle, num_sub_arrays/2);
        finalMerge(array, temp, middle+1, right, num_sub_arrays/2);
        merge(array, temp, left, right, middle);
    }
    else {
        merge(array, temp, left, right, middle);
    }
}

int main(int argc, char **argv)
{
    NUM_VALS = atoi(argv[1]);
    THREADS = atoi(argv[2]);
    BLOCKS = NUM_VALS / THREADS;

    printf("Number of threads: %d\n", THREADS);
    printf("Number of values: %d\n", NUM_VALS);
    printf("Number of blocks: %d\n", BLOCKS);

    adiak::init(NULL);
    adiak::launchdate();
    adiak::libraries();
    adiak::cmdline();
    adiak::clustername();

    std::string algorithm = "MergeSort";
    std::string programmingModel = "CUDA";
    std::string datatype = "int";
    size_t sizeOfDatatype = sizeof(int);
    std::string inputType = "Random";
    int group_number = 13;
    std::string implementation_source = "AI";

    adiak::value("Algorithm", algorithm);
    adiak::value("ProgrammingModel", programmingModel);
    adiak::value("Datatype", datatype);
    adiak::value("SizeOfDatatype", sizeOfDatatype);
    adiak::value("InputSize", NUM_VALS);
    adiak::value("InputType", inputType);
    adiak::value("num_procs", THREADS);
    adiak::value("group_num", group_number);
    adiak::value("implementation_source", implementation_source);

    CALI_MARK_BEGIN("main");

    int *d_data;
    size_t d_size = NUM_VALS * sizeof(int);

    // Generate Data
    CALI_MARK_BEGIN("data_init");
    int *h_data = (int*) malloc(NUM_VALS * sizeof(int));
    generate_data(NUM_VALS, h_data);
    CALI_MARK_END("data_init");

    // Cuda overhead
    hipMalloc((void **)&d_data, d_size);
    hipMemcpy(d_data, h_data, d_size, hipMemcpyHostToDevice);

    // Merge sort
    mergeSortKernel<<<BLOCKS, 1>>>(d_data, NUM_VALS);

    // Get data from device
    hipMemcpy(h_data, d_data, d_size, hipMemcpyDeviceToHost);

    // Final merge
    int *temp = (int*) malloc(NUM_VALS * sizeof(int));
    finalMerge(h_data, temp, 0, NUM_VALS-1, THREADS);

    // Correctness check
    CALI_MARK_BEGIN("correctness_check");
    bool correct = is_correct(NUM_VALS, h_data);
    CALI_MARK_END("correctness_check");
    std::cout << "is_correct: " << correct << "\n";
    /*
    printf("Sorted array:\n");
    for (int i = 0; i < NUM_VALS; i++) {
        printf("%d ", h_data[i]);
    }
    printf("\n");
    */

    // Clean memory
    hipFree(d_data);
    free(h_data);

    CALI_MARK_END("main");
    return 0;
}