#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <cstdint>
#include <random>
#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

// Function to generate sorted data
std::vector<int> generate_sorted_data(size_t size)
{
    std::vector<int> data(size);
    for (size_t i = 0; i < size; ++i)
    {
        data[i] = static_cast<int>(i);
    }
    return data;
}

// Function to generate reverse sorted data
std::vector<int> generate_reverse_sorted_data(size_t size)
{
    std::vector<int> data(size);
    for (size_t i = 0; i < size; ++i)
    {
        data[i] = static_cast<int>(size - i - 1);
    }
    return data;
}

// Function to generate random data
std::vector<int> generate_random_data(size_t size)
{
    std::vector<int> data(size);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<std::int64_t> dis(0, static_cast<std::int64_t>(size) * 10);

    for (size_t i = 0; i < size; ++i)
    {
        data[i] = static_cast<int>(dis(gen));
    }
    return data;
}

// Function to generate 1% perturbed data
std::vector<int> generate_perturbed_data(size_t size)
{
    std::vector<int> data = generate_sorted_data(size);
    size_t perturb_count = std::max(1UL, size / 100);

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<std::int64_t> dis(0, static_cast<std::int64_t>(size) * 10);
    std::uniform_int_distribution<size_t> index_dis(0, size - 1);

    for (size_t i = 0; i < perturb_count; ++i)
    {
        data[index_dis(gen)] = static_cast<int>(dis(gen));
    }
    return data;
}

// Check if the data is correctly sorted
bool is_correct(const std::vector<int> &data)
{
    for (size_t i = 1; i < data.size(); i++)
    {
        if (data[i - 1] > data[i])
        {
            return false;
        }
    }
    return true;
}

// CUDA kernel to pick samples from the sorted subarrays
__global__ void pick_samples(const int *data, int *samples, int stride, int num_samples)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_samples)
    {
        samples[idx] = data[idx * stride];
    }
}

// CUDA kernel to partition data based on splitters
__global__ void partition_data(int *data, int *splitters, int *buckets, int n, int num_splitters)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        int val = data[idx];
        int bucket = 0;
        while (bucket < num_splitters && val >= splitters[bucket])
        {
            bucket++;
        }
        buckets[idx] = bucket;
    }
}

// CUDA kernel to scatter elements into their correct positions
__global__ void scatter_elements(int *data, int *bucket_indices, int *bucket_start_indices, int *scattered_data, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n)
    {
        int bucket_index = bucket_indices[index];
        int pos = atomicAdd(&bucket_start_indices[bucket_index], 1);
        scattered_data[pos] = data[index];
    }
}

// CUDA kernel to count elements per bucket and initialize gather indices
__global__ void count_elements_and_prepare_gather(int *data, int *bucket_indices, int *bucket_counts, int *gather_indices, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n)
    {
        int bucket_index = bucket_indices[index];
        atomicAdd(&bucket_counts[bucket_index], 1);
        // Initialize gather_indices with -1 or a marker value to indicate unprocessed elements
        gather_indices[index] = -1;
    }
}

// Sample sort host function
void sample_sort(int *h_data, size_t size, int threadsPerBlock, int blocks)
{
    // Allocate memory and copy data to the device
    CALI_MARK_BEGIN("comm");
    CALI_MARK_BEGIN("comm_large");
    CALI_MARK_BEGIN("hipMemcpy");
    thrust::device_vector<int> d_data(h_data, h_data + size);
    CALI_MARK_END("hipMemcpy");
    CALI_MARK_END("comm_large");
    CALI_MARK_END("comm");

    // Perform local sort on the device using Thrust
    CALI_MARK_BEGIN("comp");
    CALI_MARK_BEGIN("comp_large");
    thrust::sort(d_data.begin(), d_data.end());
    CALI_MARK_END("comp_large");
    CALI_MARK_END("comp");

    // Determine the number of samples to pick
    CALI_MARK_BEGIN("comp");
    CALI_MARK_BEGIN("comp_small");
    int num_samples = sqrt(size);
    int stride = size / num_samples;
    thrust::device_vector<int> d_samples(num_samples);

    // Use a kernel to pick splitters from the sorted data
    pick_samples<<<blocks, threadsPerBlock>>>(thrust::raw_pointer_cast(d_data.data()),
                                              thrust::raw_pointer_cast(d_samples.data()), stride, num_samples);
    hipDeviceSynchronize();

    // Sort the samples on the device to get splitters
    thrust::sort(d_samples.begin(), d_samples.end());
    thrust::device_vector<int> d_splitters(num_samples - 1);
    thrust::copy(d_samples.begin() + 1, d_samples.end(), d_splitters.begin());

    // Allocate memory for buckets
    thrust::device_vector<int> d_buckets(size);

    // Partition the data into buckets according to the splitters
    partition_data<<<blocks, threadsPerBlock>>>(thrust::raw_pointer_cast(d_data.data()),
                                                thrust::raw_pointer_cast(d_splitters.data()),
                                                thrust::raw_pointer_cast(d_buckets.data()), size, num_samples - 1);
    hipDeviceSynchronize();

    // Count elements per bucket and prepare for gathering
    thrust::device_vector<int> d_bucket_counts(num_samples, 0);
    thrust::device_vector<int> d_gather_indices(size);
    count_elements_and_prepare_gather<<<blocks, threadsPerBlock>>>(thrust::raw_pointer_cast(d_data.data()),
                                                                   thrust::raw_pointer_cast(d_buckets.data()),
                                                                   thrust::raw_pointer_cast(d_bucket_counts.data()),
                                                                   thrust::raw_pointer_cast(d_gather_indices.data()),
                                                                   size);
    hipDeviceSynchronize();

    // Compute the starting indices of each bucket
    thrust::device_vector<int> d_bucket_starts(num_samples, 0);
    thrust::exclusive_scan(d_bucket_counts.begin(), d_bucket_counts.end(), d_bucket_starts.begin());
    CALI_MARK_END("comp_small");
    CALI_MARK_END("comp");

    // Scatter the elements into their correct positions
    CALI_MARK_BEGIN("comp");
    CALI_MARK_BEGIN("comp_large");
    thrust::device_vector<int> d_scattered_data(size);
    scatter_elements<<<blocks, threadsPerBlock>>>(thrust::raw_pointer_cast(d_data.data()),
                                                  thrust::raw_pointer_cast(d_buckets.data()),
                                                  thrust::raw_pointer_cast(d_bucket_starts.data()),
                                                  thrust::raw_pointer_cast(d_scattered_data.data()),
                                                  size);
    hipDeviceSynchronize();

    // Now each bucket in d_scattered_data can be sorted individually
    for (int i = 0; i < num_samples; ++i)
    {
        int start_index = d_bucket_starts[i];
        int bucket_size = (i == num_samples - 1) ? size - start_index : d_bucket_starts[i + 1] - start_index;

        if (bucket_size > 0)
        {
            thrust::sort(d_scattered_data.begin() + start_index, d_scattered_data.begin() + start_index + bucket_size);
        }
    }
    CALI_MARK_END("comp_large");
    CALI_MARK_END("comp");

    // Copy the sorted data back to host
    CALI_MARK_BEGIN("comm");
    CALI_MARK_BEGIN("comm_large");
    CALI_MARK_BEGIN("hipMemcpy");
    thrust::copy(d_scattered_data.begin(), d_scattered_data.end(), h_data);
    CALI_MARK_END("hipMemcpy");
    CALI_MARK_END("comm_large");
    CALI_MARK_END("comm");
}

int main(int argc, char **argv)
{
    // Check for correct number of arguments
    if (argc < 4)
    {
        std::cerr << "Usage: " << argv[0] << " <inputType> <size> <numThreads>\n";
        return 1;
    }

    // Parse the command line arguments
    std::string inputType = argv[1];
    size_t size = std::stoul(argv[2]);
    int numThreads = std::stoi(argv[3]);

    int threadsPerBlock = 256;
    int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;

    adiak::init(NULL);
    adiak::launchdate();
    adiak::libraries();
    adiak::cmdline();
    adiak::clustername();

    std::string algorithm = "SampleSort";
    std::string programmingModel = "CUDA";
    std::string datatype = "int";
    size_t sizeOfDatatype = sizeof(int);
    int group_number = 13;
    std::string implementation_source = "AI";

    adiak::value("Algorithm", algorithm);
    adiak::value("ProgrammingModel", programmingModel);
    adiak::value("Datatype", datatype);
    adiak::value("SizeOfDatatype", sizeOfDatatype);
    adiak::value("InputSize", size);
    adiak::value("InputType", inputType);
    adiak::value("num_threads", numThreads);
    adiak::value("num_blocks", numBlocks);
    adiak::value("group_num", group_number);
    adiak::value("implementation_source", implementation_source);

    CALI_MARK_BEGIN("main");

    CALI_MARK_BEGIN("data_init");
    std::vector<int> data;
    if (inputType == "Sorted")
    {
        data = generate_sorted_data(size);
    }
    else if (inputType == "ReverseSorted")
    {
        data = generate_reverse_sorted_data(size);
    }
    else if (inputType == "Random")
    {
        data = generate_random_data(size);
    }
    else if (inputType == "1%perturbed")
    {
        data = generate_perturbed_data(size);
    }
    else
    {
        std::cerr << "Invalid input type. Use 'Sorted', 'ReverseSorted', 'Random', or '1%perturbed'.\n";
        return 1;
    }
    CALI_MARK_END("data_init");

    sample_sort(data.data(), data.size(), threadsPerBlock, numBlocks);

    CALI_MARK_BEGIN("correctness_check");
    bool correct = is_correct(data);
    if (!correct)
    {
        std::cerr << "Error: The algorithm did not sort the data correctly." << std::endl;
    }
    CALI_MARK_END("correctness_check");

    CALI_MARK_END("main");
    return 0;
}
