#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <cstdint>
#include <random>
#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <thrust/count.h>
#include <thrust/transform.h>
#include <thrust/gather.h>
#include <thrust/scatter.h>
#include <thrust/iterator/counting_iterator.h>
#include <cmath>

// Global variables
int THREADS;
int BLOCKS;
size_t NUM_VALS;

// Function to generate sorted data
std::vector<int> generate_sorted_data()
{
    std::vector<int> data(NUM_VALS);
    for (size_t i = 0; i < NUM_VALS; ++i)
    {
        data[i] = static_cast<int>(i);
    }
    return data;
}

// Function to generate reverse sorted data
std::vector<int> generate_reverse_sorted_data()
{
    std::vector<int> data(NUM_VALS);
    for (size_t i = 0; i < NUM_VALS; ++i)
    {
        data[i] = static_cast<int>(NUM_VALS - i - 1);
    }
    return data;
}

// Function to generate random data
std::vector<int> generate_random_data()
{
    std::vector<int> data(NUM_VALS);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<std::int64_t> dis(0, static_cast<std::int64_t>(NUM_VALS) * 10);

    for (size_t i = 0; i < NUM_VALS; ++i)
    {
        data[i] = static_cast<int>(dis(gen));
    }
    return data;
}

// Function to generate 1% perturbed data
std::vector<int> generate_perturbed_data()
{
    std::vector<int> data = generate_sorted_data();
    size_t perturb_count = std::max(1UL, NUM_VALS / 100);

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<std::int64_t> dis(0, static_cast<std::int64_t>(NUM_VALS) * 10);
    std::uniform_int_distribution<size_t> index_dis(0, NUM_VALS - 1);

    for (size_t i = 0; i < perturb_count; ++i)
    {
        data[index_dis(gen)] = static_cast<int>(dis(gen));
    }
    return data;
}

// Check if the data is correctly sorted
bool is_correct(const std::vector<int> &data)
{
    for (size_t i = 1; i < data.size(); i++)
    {
        if (data[i - 1] > data[i])
        {
            return false;
        }
    }
    return true;
}

/**
 * GPU Sample Sort
 * -----------------------
 * Copyright (c) 2009-2019 Nikolaj Leischner and Vitaly Osipov
 *
 * Permission is hereby granted, free of charge, to any person
 * obtaining a copy of this software and associated documentation
 * files (the "Software"), to deal in the Software without
 * restriction, including without limitation the rights to use,
 * copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following
 * conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
 * OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
 * HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
 * WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
 * OTHER DEALINGS IN THE SOFTWARE.
 **/

#include <algorithm>
#include <stack>
#include <vector>
#include <queue>
#include <random>
#include <limits>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include "detail/constants.h"
#include "detail/bucket.h"
#include "detail/create_bst.h"
#include "detail/find_buckets.h"
#include "detail/scatter.h"
#include "detail/quicksort.h"
#include "detail/copy_buckets.h"
#include "detail/temporary_device_memory.h"

namespace SampleSort
{

    int clamp(int value, int lo, int hi)
    {
        return std::max(lo, std::min(value, hi));
    }

    template <int COPY_THREADS, int MAX_BLOCK_COUNT, bool KEYS_ONLY, typename KeyType, typename ValueType>
    void move_to_output(std::priority_queue<Bucket> &swapped_buckets, KeyType *keys,
                        const TemporaryDeviceMemory<KeyType> &keys_buffer, ValueType *values,
                        const TemporaryDeviceMemory<ValueType> &values_buffer)
    {
        int batch_size = static_cast<int>(std::min(swapped_buckets.size(), static_cast<size_t>(MAX_BLOCK_COUNT)));
        TemporaryDeviceMemory<Bucket> dev_swapped_bucket_data((size_t)batch_size);
        std::vector<Bucket> swapped_bucket_data;

        while (!swapped_buckets.empty())
        {
            swapped_bucket_data.clear();
            batch_size = static_cast<int>(std::min(swapped_buckets.size(), static_cast<size_t>(MAX_BLOCK_COUNT)));

            for (int i = 0; i < batch_size; ++i)
            {
                swapped_bucket_data.push_back(swapped_buckets.top());
                swapped_buckets.pop();
            }

            dev_swapped_bucket_data.copy_to_device(swapped_bucket_data.data());

            if (KEYS_ONLY)
                copy_buckets<COPY_THREADS><<<batch_size, COPY_THREADS>>>(keys, keys_buffer.data, dev_swapped_bucket_data.data);
            else
                copy_buckets<COPY_THREADS><<<batch_size, COPY_THREADS>>>(keys, keys_buffer.data, values, values_buffer.data, dev_swapped_bucket_data.data);
        }
    }

    template <bool KEYS_ONLY, typename KeyType, typename ValueType, typename CompType>
    void sort_buckets(std::priority_queue<Bucket> &small_buckets, KeyType *keys,
                      const TemporaryDeviceMemory<KeyType> &keys_buffer, ValueType *values,
                      const TemporaryDeviceMemory<ValueType> &values_buffer, CompType comp,
                      int sort_threads, int max_block_count)
    {
        // Below this size odd-even-merge-sort is used in the CTA sort.
        const unsigned int LOCAL_SORT_SIZE = 2048;
        // Might want to choose a different size for key-value sorting, since the
        // shared memory requirements are higher.
        const unsigned int LOCAL_SORT_SIZE_KV = 2048;
        int batch_size = static_cast<int>(std::min(small_buckets.size(), static_cast<size_t>(MAX_BLOCK_COUNT)));
        TemporaryDeviceMemory<Bucket> dev_small_bucket_data((size_t)batch_size);
        std::vector<Bucket> small_bucket_data;

        while (!small_buckets.empty())
        {
            small_bucket_data.clear();
            batch_size = static_cast<int>(std::min(small_buckets.size(), static_cast<size_t>(MAX_BLOCK_COUNT)));

            for (int i = 0; i < batch_size; ++i)
            {
                small_bucket_data.push_back(small_buckets.top());
                small_buckets.pop();
            }

            dev_small_bucket_data.copy_to_device(small_bucket_data.data());

            if (KEYS_ONLY)
                quicksort<LOCAL_SORT_SIZE, sort_threads><<<batch_size, sort_threads>>>(keys, keys_buffer.data, dev_small_bucket_data.data, comp);
            else
                quicksort<LOCAL_SORT_SIZE_KV, sort_threads><<<batch_size, sort_threads>>>(keys, keys_buffer.data, values, values_buffer.data, dev_small_bucket_data.data, comp);
        }
    }

    template <typename KeyPtrType, typename ValuePtrType, typename StrictWeakOrdering, bool KEYS_ONLY>
    void sort(KeyPtrType begin, KeyPtrType end, ValuePtrType values_begin, StrictWeakOrdering comp)
    {
        int sort_threads = THREADS;
        int max_block_count = BLOCKS;

        const int A = 32;
        // Smaller oversampling factor, used when all buckets are smaller than some size.
        const int SMALL_A = A / 2;
        // How large should the largest bucket be to allow using the smaller oversampling factor?
        const int REDUCED_OVERSAMPLING_LIMIT = 1 << 25;
        // Number of replicated bucket counters per thread block in the bucket finding / scattering kernels.
        const int COUNTERS = 1;
        // Factor for additional counter replication in the bucket finding kernel.
        const int COUNTER_COPIES = 1;

        const int LOCAL_SORT_SIZE = 2048;
        const int BST_THREADS = 128;
        const int FIND_THREADS = 128;
        const int SCATTER_THREADS = 128;
        // Must be a power of 2.
        const int LOCAL_THREADS = 256;
        const int COPY_THREADS = 128;

        // The number of elements/thread is chosen so that at least this many CTAs are used, if possible.
        const int DESIRED_CTA_COUNT = 1024;

        const int MAX_BLOCK_COUNT = (1 << 29) - 1;

        typedef typename thrust::iterator_traits<KeyPtrType>::value_type KeyType;
        typedef typename thrust::iterator_traits<ValuePtrType>::value_type ValueType;
        typedef StrictWeakOrdering CompType;

        KeyType *keys = thrust::raw_pointer_cast(&*begin);
        ValueType *values = thrust::raw_pointer_cast(&*values_begin);

        const int size = static_cast<int>(end - begin);
        if (size == 0)
            return;

        const int block_sort_limit = clamp(static_cast<int>(size / (2 * std::sqrt(static_cast<float>(K)))), 1 << 14,
                                           1 << 18);

        std::stack<Bucket> large_buckets;
        // Buckets are ordered by size, which improves the performance of the
        // CTA level sorting. Helps the gpu's scheduler?
        std::priority_queue<Bucket> small_buckets;
        std::priority_queue<Bucket> swapped_buckets;

        // Push the whole input on a stack.
        Bucket init(0, size);

        if (size < block_sort_limit)
            small_buckets.push(init);
        else
            large_buckets.push(init);

        TemporaryDeviceMemory<KeyType> keys_buffer(size);

        std::random_device rd;
        std::mt19937 gen(rd());
        // Seeded with a constant value for reproducible benchmark results.
        gen.seed(17);
        std::uniform_int_distribution<int> distribution;
        auto *rng = new Lrand48();

        TemporaryDeviceMemory<ValueType> values_buffer(KEYS_ONLY ? size : 0);

        // Cooperatively k-way split large buckets. Search tree creation is done for several large buckets in parallel.
        while (!large_buckets.empty())
        {
            // Grab as many large buckets as possible, within the CTA count limitation for a kernel call.
            std::vector<Bucket> buckets;
            int max_blocks_per_bucket = 0;
            while (!large_buckets.empty() && buckets.size() < MAX_BLOCK_COUNT)
            {
                Bucket b = large_buckets.top();
                // Adjust the number of elements/thread according to the bucket size.
                int keys_per_thread =
                    static_cast<int>(std::max(1, static_cast<int>(ceil(
                                                     static_cast<double>(b.size) / (DESIRED_CTA_COUNT * FIND_THREADS)))));
                int block_count =
                    static_cast<int>(ceil((static_cast<double>(b.size) / (keys_per_thread * FIND_THREADS))));

                b.keys_per_thread = keys_per_thread;
                max_blocks_per_bucket = std::max(max_blocks_per_bucket, block_count);
                buckets.push_back(b);
                large_buckets.pop();
            }

            // Copy bucket parameters to the GPU.
            TemporaryDeviceMemory<Bucket> dev_bucketParams(buckets.size());
            dev_bucketParams.copy_to_device(buckets.data());

            // Create the binary search trees.
            TemporaryDeviceMemory<KeyType> bst(K * buckets.size());

            rng->init(static_cast<int>((buckets.size() * BST_THREADS)), distribution(gen));

            const int bst_blocks = static_cast<int>(buckets.size());

            // One CTA creates the search tree for one bucket. In the first step only
            // one multiprocessor will be occupied. If no bucket is larger than a certain size,
            // use less oversampling.
            if (block_sort_limit < REDUCED_OVERSAMPLING_LIMIT)
            {
                TemporaryDeviceMemory<KeyType> sample(SMALL_A * K * buckets.size());
                TemporaryDeviceMemory<KeyType> sample_buffer(SMALL_A * K * buckets.size());
                create_bst<K, SMALL_A, BST_THREADS, LOCAL_SORT_SIZE><<<bst_blocks, BST_THREADS>>>(keys, keys_buffer.data, dev_bucketParams.data, bst.data, sample.data, sample_buffer.data, *rng,
                                                                                                  comp);
            }
            else
            {
                TemporaryDeviceMemory<KeyType> sample(A * K * buckets.size());
                TemporaryDeviceMemory<KeyType> sample_buffer(A * K * buckets.size());
                create_bst<K, A, BST_THREADS, LOCAL_SORT_SIZE><<<bst_blocks, BST_THREADS>>>(keys, keys_buffer.data, dev_bucketParams.data, bst.data, sample.data, sample_buffer.data, *rng,
                                                                                            comp);
            }

            rng->destroy();

            // Fetch the bucket parameters again which now contain information about which buckets
            // have only equal splitters. Would be sufficient to just fetch an array of bool flags instead
            // of all parameters. But from profiling it looks as if that would be over-optimization.
            dev_bucketParams.copy_to_host(buckets.data());

            TemporaryDeviceMemory<int> dev_bucket_counters(static_cast<size_t>(K * COUNTERS * max_blocks_per_bucket));

            std::vector<int> new_bucket_bounds(K * buckets.size());
            TemporaryDeviceMemory<int> dev_new_bucket_bounds(K * buckets.size());

            // Loop over the large buckets. The limit for considering a bucket to be large should ensure
            // that the bucket-finding and scattering kernels are launched with a sufficient number of CTAs
            // to make use of all available multiprocessors.
            for (int i = 0; i < buckets.size(); ++i)
            {
                Bucket b = buckets[i];

                int block_count = static_cast<int>(ceil(
                    static_cast<double>(b.size) / (FIND_THREADS * b.keys_per_thread)));

                int from = b.start;
                int to = b.start + b.size;

                KeyType *input = b.flipped ? keys_buffer.data : keys;
                KeyType *output = b.flipped ? keys : keys_buffer.data;
                ValueType *values_input = b.flipped ? values_buffer.data : values;
                ValueType *values_output = b.flipped ? values : values_buffer.data;

                hipMemcpyToSymbol(HIP_SYMBOL(bst_cache), bst.data + K * i, K * sizeof(KeyType), 0, hipMemcpyDeviceToDevice);

                // If all keys in the sample are equal, check if the whole bucket contains only one key.
                if (b.degenerated)
                {
                    thrust::device_ptr<KeyType> dev_input(input + from);
                    KeyType min_key, max_key;
                    hipMemcpy(&min_key, thrust::min_element(dev_input, dev_input + b.size).get(), sizeof(KeyType),
                               hipMemcpyDeviceToHost);
                    hipMemcpy(&max_key, thrust::max_element(dev_input, dev_input + b.size).get(), sizeof(KeyType),
                               hipMemcpyDeviceToHost);

                    if (!comp(min_key, max_key) && !comp(max_key, min_key))
                    {
                        buckets[i].constant = true;
                        // Skip the rest, the bucket is already sorted.
                        continue;
                    }
                }

                // Find buckets.
                if (!b.degenerated)
                    find_buckets<K, LOG_K, FIND_THREADS, COUNTERS, COUNTER_COPIES, false>
                        <<<block_count, FIND_THREADS>>>(input, from, to, dev_bucket_counters.data, b.keys_per_thread, comp);
                else
                    find_buckets<K, LOG_K, FIND_THREADS, COUNTERS, COUNTER_COPIES, true>
                        <<<block_count, FIND_THREADS>>>(input, from, to, dev_bucket_counters.data, b.keys_per_thread, comp);

                // Scan over the bucket counters, yielding the array positions the blocks of the scattering kernel need to write to.
                thrust::device_ptr<int> dev_counters(dev_bucket_counters.data);
                thrust::inclusive_scan(dev_counters, dev_counters + K * COUNTERS * block_count, dev_counters);

                if (KEYS_ONLY)
                {
                    if (!b.degenerated)
                        scatter<K, LOG_K, FIND_THREADS, SCATTER_THREADS, COUNTERS, false>
                            <<<block_count, SCATTER_THREADS>>>(input, from, to, output, dev_bucket_counters.data,
                                                               dev_new_bucket_bounds.data + K * i, b.keys_per_thread, comp);
                    else
                        scatter<K, LOG_K, FIND_THREADS, SCATTER_THREADS, COUNTERS, true>
                            <<<block_count, SCATTER_THREADS>>>(input, from, to, output, dev_bucket_counters.data,
                                                               dev_new_bucket_bounds.data + K * i, b.keys_per_thread,
                                                               comp);
                }
                else
                {
                    if (!b.degenerated)
                        scatter<K, LOG_K, FIND_THREADS, SCATTER_THREADS, COUNTERS, false>
                            <<<block_count, SCATTER_THREADS>>>(input, values_input, from, to, output, values_output,
                                                               dev_bucket_counters.data, dev_new_bucket_bounds.data + K * i,
                                                               b.keys_per_thread, comp);
                    else
                        scatter<K, LOG_K, FIND_THREADS, SCATTER_THREADS, COUNTERS, true>
                            <<<block_count, SCATTER_THREADS>>>(input, values_input, from, to, output, values_output,
                                                               dev_bucket_counters.data, dev_new_bucket_bounds.data + K * i,
                                                               b.keys_per_thread, comp);
                }
            }

            dev_new_bucket_bounds.copy_to_host(new_bucket_bounds.data());

            for (int i = 0; i < buckets.size(); i++)
            {
                if (!buckets[i].degenerated)
                {
                    for (int j = 0; j < K; j++)
                    {
                        int start = (j > 0) ? new_bucket_bounds[K * i + j - 1] : buckets[i].start;
                        int bucket_size = new_bucket_bounds[K * i + j] - start;
                        Bucket new_bucket(start, bucket_size, !buckets[i].flipped);

                        // Depending on it's size push the bucket on a different stack.
                        if (new_bucket.size > block_sort_limit)
                            large_buckets.push(new_bucket);
                        else if (new_bucket.size > 1)
                            small_buckets.push(new_bucket);
                        else if (new_bucket.size == 1 && new_bucket.flipped)
                            swapped_buckets.push(new_bucket);
                    }
                }
                else if (!buckets[i].constant)
                {
                    // There are only 3 buckets if all splitters were equal.
                    for (int j = 0; j < 3; j++)
                    {
                        int start = (j > 0) ? new_bucket_bounds[K * i + j - 1] : buckets[i].start;
                        int bucket_size = new_bucket_bounds[K * i + j] - start;
                        Bucket new_bucket(start, bucket_size, !buckets[i].flipped);

                        // Bucket with id 1 contains only equal keys, there is no need to sort it.
                        if (j == 1)
                        {
                            if (new_bucket.flipped)
                                swapped_buckets.push(new_bucket);
                        }
                        else if (new_bucket.size > block_sort_limit)
                            large_buckets.push(new_bucket);
                        else if (new_bucket.size > 1)
                            small_buckets.push(new_bucket);
                        else if (new_bucket.size == 1 && new_bucket.flipped)
                            swapped_buckets.push(new_bucket);
                    }
                }
                else
                {
                    // The bucket only contains equal keys. No need for sorting.
                    if (buckets[i].flipped)
                        swapped_buckets.push(buckets[i]);
                }
            }
        }
        delete rng;

        CALI_MARK_BEGIN("comp_small");
        move_to_output<COPY_THREADS, MAX_BLOCK_COUNT, KEYS_ONLY>(swapped_buckets, keys, keys_buffer, values, values_buffer);
        CALI_MARK_END("comp_small");

        CALI_MARK_BEGIN("comp_large");
        sort_buckets<KEYS_ONLY>(small_buckets, keys, keys_buffer, values, values_buffer, comp, sort_threads, max_block_count);
        CALI_MARK_END("comp_large");
    }

    void sort_by_key(std::uint16_t *keys, std::uint16_t *keys_end, std::uint64_t *values)
    {
        SampleSort::sort<std::uint16_t *, std::uint64_t *, thrust::less<std::uint16_t>, false>(keys, keys_end, values, thrust::less<std::uint16_t>());
    }

    void sort_by_key(std::uint32_t *keys, std::uint32_t *keys_end, std::uint64_t *values)
    {
        SampleSort::sort<std::uint32_t *, std::uint64_t *, thrust::less<std::uint32_t>, false>(keys, keys_end, values, thrust::less<std::uint32_t>());
    }

    void sort_by_key(std::uint64_t *keys, std::uint64_t *keys_end, std::uint64_t *values)
    {
        SampleSort::sort<std::uint64_t *, std::uint64_t *, thrust::less<std::uint64_t>, false>(keys, keys_end, values, thrust::less<std::uint64_t>());
    }

    void sort(std::uint16_t *keys, std::uint16_t *keys_end)
    {
        SampleSort::sort<std::uint16_t *, std::uint16_t *, thrust::less<std::uint16_t>, true>(keys, keys_end, 0, thrust::less<std::uint16_t>());
    }

    void sort(std::uint32_t *keys, std::uint32_t *keys_end)
    {
        SampleSort::sort<std::uint32_t *, std::uint32_t *, thrust::less<std::uint32_t>, true>(keys, keys_end, 0, thrust::less<std::uint32_t>());
    }

    void sort(std::uint64_t *keys, std::uint64_t *keys_end)
    {
        SampleSort::sort<std::uint64_t *, std::uint64_t *, thrust::less<std::uint64_t>, true>(keys, keys_end, 0, thrust::less<std::uint64_t>());
    }
}

int main(int argc, char **argv)
{
    CALI_MARK_BEGIN("main");

    // Check for correct number of arguments
    if (argc < 4)
    {
        std::cerr << "Usage: " << argv[0] << " <size> <numThreads> <inputType>\n";
        return 1;
    }

    // Parse the command line arguments
    NUM_VALS = std::stoul(argv[1]);
    THREADS = std::stoi(argv[2]);
    BLOCKS = (NUM_VALS + THREADS - 1) / THREADS;
    std::string inputTypeShort = argv[3];

    std::string inputType;
    if (inputTypeShort == "r")
    {
        inputType = "Random";
    }
    else if (inputTypeShort == "s")
    {
        inputType = "Sorted";
    }
    else if (inputTypeShort == "rs")
    {
        inputType = "Reverse Sorted";
    }
    else if (inputTypeShort == "p")
    {
        inputType = "1% Perturbed";
    }
    else
    {
        std::cerr << "Invalid input type. Use 'r', 's', 'rs', or 'p'.\n";
        return 1;
    }

    adiak::init(NULL);
    adiak::launchdate();
    adiak::libraries();
    adiak::cmdline();
    adiak::clustername();

    std::string algorithm = "SampleSort";
    std::string programmingModel = "CUDA";
    std::string datatype = "int";
    size_t sizeOfDatatype = sizeof(int);
    int group_number = 13;
    std::string implementation_source = "Online";

    adiak::value("Algorithm", algorithm);
    adiak::value("ProgrammingModel", programmingModel);
    adiak::value("Datatype", datatype);
    adiak::value("SizeOfDatatype", sizeOfDatatype);
    adiak::value("InputSize", NUM_VALS);
    adiak::value("InputType", inputType);
    adiak::value("num_threads", THREADS);
    adiak::value("num_blocks", BLOCKS);
    adiak::value("group_num", group_number);
    adiak::value("implementation_source", implementation_source);

    CALI_MARK_BEGIN("data_init");
    std::vector<int> data;
    if (inputType == "Sorted")
    {
        data = generate_sorted_data();
    }
    else if (inputType == "Reverse Sorted")
    {
        data = generate_reverse_sorted_data();
    }
    else if (inputType == "Random")
    {
        data = generate_random_data();
    }
    else if (inputType == "1% Perturbed")
    {
        data = generate_perturbed_data();
    }
    else
    {
        std::cerr << "Invalid input type. Use 'Sorted', 'Reverse Sorted', 'Random', or '1% Perturbed'.\n";
        return 1;
    }
    CALI_MARK_END("data_init");

    CALI_MARK_BEGIN("comm");
    CALI_MARK_BEGIN("comm_small");
    CALI_MARK_END("comm_small");
    CALI_MARK_END("comm");

    CALI_MARK_BEGIN("comm");
    CALI_MARK_BEGIN("comm_large");
    CALI_MARK_BEGIN("hipMemcpy");
    thrust::device_vector<std::uint32_t> d_data(data.begin(), data.end());
    CALI_MARK_END("hipMemcpy");
    CALI_MARK_END("comm_large");
    CALI_MARK_END("comm");

    SampleSort::sort(d_data.data().get(), d_data.data().get() + d_data.size(), thrust::less<std::uint32_t>());

    CALI_MARK_BEGIN("comm");
    CALI_MARK_BEGIN("comm_large");
    CALI_MARK_BEGIN("hipMemcpy");
    thrust::copy(d_data.begin(), d_data.end(), data.begin());
    CALI_MARK_END("hipMemcpy");
    CALI_MARK_END("comm_large");
    CALI_MARK_END("comm");

    CALI_MARK_BEGIN("correctness_check");
    bool correct = is_correct(data);
    if (!correct)
    {
        std::cerr << "Error: The algorithm did not sort the data correctly." << std::endl;
    }
    CALI_MARK_END("correctness_check");

    CALI_MARK_END("main");
    return 0;
}
