#include "hip/hip_runtime.h"
/*
 * Parallel bitonic sort using CUDA.
 * Compile with
 * nvcc bitonic_sort.cu
 * Based on http://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm
 * License: BSD 3
 */

#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int THREADS;
int BLOCKS;
int NUM_VALS;

float effective_bandwidth_gb_s = 0.0;
float bitonic_sort_step_time = 0.0;
float cudaMemcpy_host_to_device_time = 0.0;
float cudaMemcpy_device_to_host_time = 0.0;

//const char* bitonic_sort_step_region = "bitonic_sort_step";
//const char* cudaMemcpy_host_to_device = "cudaMemcpy_host_to_device";
//const char* cudaMemcpy_device_to_host = "cudaMemcpy_device_to_host";

const char* comm_large = "comm_large";
const char* comm = "comm";
const char* hipMemcpy = "hipMemcpy";
const char* comp = "comp";
const char* comp_large = "comp_large";
const char* correctness_check = "correctness_check";
const char* main = "main";

void print_elapsed(clock_t start, clock_t stop)
{
  double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Elapsed time: %.3fs\n", elapsed);
}

float random_float()
{
  return (float)rand()/(float)RAND_MAX;
}

void array_print(float *arr, int length) 
{
  int i;
  for (i = 0; i < length; ++i) {
    printf("%1.3f ",  arr[i]);
  }
  printf("\n");
}

void array_fill(float *arr, int length)
{
  srand(time(NULL));
  int i;
  for (i = 0; i < length; ++i) {
    arr[i] = random_float();
  }
}

__global__ void bitonic_sort_step(float *dev_values, int j, int k)
{
  unsigned int i, ixj; /* Sorting partners: i and ixj */
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  /* The threads with the lowest ids sort the array. */
  if ((ixj)>i) {
    if ((i&k)==0) {
      /* Sort ascending */
      if (dev_values[i]>dev_values[ixj]) {
        /* exchange(i,ixj); */
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
    if ((i&k)!=0) {
      /* Sort descending */
      if (dev_values[i]<dev_values[ixj]) {
        /* exchange(i,ixj); */
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
  }
}

/**
 * Inplace bitonic sort using CUDA.
 */
void bitonic_sort(float *values)
{

  float *dev_values;
  size_t size = NUM_VALS * sizeof(float);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  hipMalloc((void**) &dev_values, size);
  
  //MEM COPY FROM HOST TO DEVICE
  CALI_MARK_BEGIN(comm);
  CALI_MARK_BEGIN(comm_large);
  //hipEventRecord(start);
  hipMemcpy(dev_values, values, size, comm_large);
  //hipEventRecord(stop);
  //hipEventSynchronize(stop);
  //hipEventElapsedTime(&cudaMemcpy_host_to_device_time, start, stop);
  CALI_MARK_END(comm_large);
  CALI_MARK_END(comm);

  dim3 blocks(BLOCKS,1);    /* Number of blocks   */
  dim3 threads(THREADS,1);  /* Number of threads  */
  
  CALI_MARK_BEGIN(comp);
  CALI_MARK_BEGIN(comp_large);
  hipEventRecord(start);
  int j, k;
  int calls = 0;
  
  /* Major step */
  for (k = 2; k <= NUM_VALS; k <<= 1) {
    /* Minor step */
    for (j=k>>1; j>0; j=j>>1) {
      calls++;
      bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
    }
  }
  //hipDeviceSynchronize();
  //hipEventRecord(stop);
  //hipEventSynchronize(stop);
  //hipEventElapsedTime(&bitonic_sort_step_time, start, stop);
  CALI_MARK_END(comp_large);
  CALI_MARK_END(comp);
  
  //MEM COPY FROM DEVICE TO HOST
  CALI_MARK_BEGIN(cudaMemcpy_device_to_host);
  //hipEventRecord(start);
  hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
  //hipEventRecord(stop);
  //hipEventSynchronize(stop);
  //hipEventElapsedTime(&cudaMemcpy_device_to_host_time, start, stop);
  CALI_MARK_END(cudaMemcpy_device_to_host);
  
  
  //double sizeGB = size * calls * 6;
  //effective_bandwidth_gb_s = (sizeGB*1e-9)/(bitonic_sort_step_time/1000);
  
  //effective_bandwidth_gb_s = (NUM_VALS*4*2) / bitonic_sort_step_region;
  
  hipFree(dev_values);
  
}

int main(int argc, char *argv[])
{
  THREADS = atoi(argv[1]);
  NUM_VALS = atoi(argv[2]);
  BLOCKS = NUM_VALS / THREADS;

  printf("Number of threads: %d\n", THREADS);
  printf("Number of values: %d\n", NUM_VALS);
  printf("Number of blocks: %d\n", BLOCKS);

  // Create caliper ConfigManager object
  cali::ConfigManager mgr;
  mgr.start();

  clock_t start, stop;
  
  CALI_MARK_BEGIN(main);

  CALI_MARK_BEGIN(data_init);
  float *values = (float*) malloc( NUM_VALS * sizeof(float));
  array_fill(values, NUM_VALS);
  CALI_MARK_END(data_init);

  start = clock();
  bitonic_sort(values); /* Inplace */
  stop = clock();

  print_elapsed(start, stop);
  
  CALI_MARK_END(main);
  
  

  // Store results in these variables.
  //float effective_bandwidth_gb_s;
  //float bitonic_sort_step_time;
  //float cudaMemcpy_host_to_device_time;
  //float cudaMemcpy_device_to_host_time;

  adiak::init(NULL);
  adiak::user();
  adiak::launchdate();
  adiak::libraries();
  adiak::cmdline();
  adiak::clustername();
  adiak::value("num_threads", THREADS);
  adiak::value("num_blocks", BLOCKS);
  adiak::value("num_vals", NUM_VALS);
  adiak::value("program_name", "cuda_bitonic_sort");
  adiak::value("datatype_size", sizeof(float));
  adiak::value("effective_bandwidth (GB/s)", effective_bandwidth_gb_s);
  adiak::value("bitonic_sort_step_time", bitonic_sort_step_time);
  adiak::value("cudaMemcpy_host_to_device_time", cudaMemcpy_host_to_device_time);
  adiak::value("cudaMemcpy_device_to_host_time", cudaMemcpy_device_to_host_time);

  // Flush Caliper output before finalizing MPI
  mgr.stop();
  mgr.flush();
  
  printf("Effective Bandwidth: %.5f\n", effective_bandwidth_gb_s);
}
